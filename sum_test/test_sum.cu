#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void cu_add(int* a, int* b, int* sum) {
    *sum = *a + *b;
}


void c_add(int a, int b, int* sum) {
    *sum = a + b;
}

void cu_add(int a, int b, int* sum) {

    int *d_a, *d_b, *d_sum;

    int size = sizeof(int);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_sum, size);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    cuda_add<<<1, 1>>>(d_a, d_b, d_sum);
    hipMemcpy(sum, d_sum, size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_sum);

    printf("Cuda result: %d\n", *sum);
}